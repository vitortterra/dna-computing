
#include <hip/hip_runtime.h>
#include <sys/time.h>

double wall_time()
{
  struct timeval tv;
  struct timezone tz;

  gettimeofday(&tv, &tz);
  return(tv.tv_sec + tv.tv_usec/1000000.0);
}
