#include "hip/hip_runtime.h"
#include "dna.h"

#define THREADS_PER_BLOCK 1024

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if(hipSuccess != err) {
    fprintf(stderr, "checkCudaErrors() Driver API error = %04d \"%s\" from file <%s>, line %i.\n",
            err, hipGetErrorString(err), file, line );
    exit(EXIT_FAILURE);
  }
}

uint8_t *length_h, *length_d;
char *strands_h, *strands_d;
int *tubeID_h, *tubeID_d; 
long strandCount;
size_t lengthSize = (size_t) MAX_STRAND_COUNT * sizeof(uint8_t);
size_t tubeIDSize = (size_t) MAX_STRAND_COUNT * sizeof(int);
size_t strandsSize = (size_t) MAX_STRAND_COUNT * (size_t) MAX_STRAND_LENGTH * sizeof(char);


void DNA_init() {
    length_h = (uint8_t *)calloc((size_t) MAX_STRAND_COUNT, sizeof(uint8_t));
    tubeID_h = (int *)malloc((size_t) MAX_STRAND_COUNT * sizeof(int));
    memset(tubeID_h, -1, (size_t) MAX_STRAND_COUNT * sizeof(int));    
    strands_h = (char *)calloc((size_t) MAX_STRAND_COUNT * (size_t) MAX_STRAND_LENGTH, sizeof(char));

    strandCount = 0;

    checkCudaErrors(hipMalloc((void **) &length_d, lengthSize));
    checkCudaErrors(hipMemset(length_d, 0, lengthSize));

    checkCudaErrors(hipMalloc((void **) &tubeID_d, tubeIDSize));
    checkCudaErrors(hipMemset(tubeID_d, -1, tubeIDSize));

    checkCudaErrors(hipMalloc((void **) &strands_d, strandsSize));
    checkCudaErrors(hipMemset(strands_d, 0, strandsSize));
}

void DNA_finalize() {
    checkCudaErrors(hipFree(length_d));
    checkCudaErrors(hipFree(tubeID_d));
    checkCudaErrors(hipFree(strands_d));
    checkCudaErrors(hipDeviceReset());
    free(length_h);
    free(tubeID_h);
    free(strands_h);
}

void showActiveStrands() {
    int strandID, i;

    checkCudaErrors(hipMemcpy(length_h, length_d, lengthSize, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(tubeID_h, tubeID_d, tubeIDSize, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(strands_h, strands_d, strandsSize, hipMemcpyDeviceToHost));

    printf("Active strands: \n");
    for (strandID = 0; strandID < strandCount; strandID++) {
        if (tubeID_h[strandID] >= 0) {
            printf("%d, tube %d: \n", strandID, tubeID_h[strandID]);
            for (i = 0; i < length_h[strandID]; i++)
                printf("%d ", strands_h[ithBit(strandID, i)]);
            printf("\n\n");
        }
    }
    printf("------------------\n");
}

char* getStrand(int strandID, uint8_t * len) {
    char* strand;

    checkCudaErrors(hipMemcpy(len, &length_d[strandID], sizeof(uint8_t), hipMemcpyDeviceToHost));
    strand = (char*)malloc(*len * sizeof(char));    
    checkCudaErrors(hipMemcpy(strand, &strands_d[ithBit(strandID, 0)], *len * sizeof(char), hipMemcpyDeviceToHost));

    return strand;
}

int getStrandTubeID(int strandID) {
    int solutionTubeID;

    checkCudaErrors(hipMemcpy(&solutionTubeID, &tubeID_d[strandID], sizeof(int), hipMemcpyDeviceToHost));
    return solutionTubeID;
}

__device__ int nextStrandID_d;

__global__ void amplifyKernel(int T0, int T1, int strandCount, 
    uint8_t * length_d, int * tubeID_d, char * strands_d) {
    
    int strandID = blockIdx.x * blockDim.x + threadIdx.x;
    int localNextStrandID;
    uint8_t len;

    if (strandID < strandCount) {
        if (tubeID_d[strandID] == T0) {
            localNextStrandID = atomicAdd(&nextStrandID_d, 1);
            len = length_d[strandID];
            tubeID_d[localNextStrandID] = T1;
            length_d[localNextStrandID] = len;
            memcpy(&strands_d[ithBit(localNextStrandID, 0)], &strands_d[ithBit(strandID, 0)], len * sizeof(char));
        }
    }
}

void amplify(int T0, int T1) {
    int numBlocks = strandCount/THREADS_PER_BLOCK + 1;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nextStrandID_d), &strandCount, sizeof(int), 0, hipMemcpyHostToDevice));
    amplifyKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
        T0,
        T1,
        strandCount,
        length_d, 
        tubeID_d,
        strands_d);
    checkCudaErrors(hipMemcpyFromSymbol(&strandCount, HIP_SYMBOL(nextStrandID_d), sizeof(int), 0, hipMemcpyDeviceToHost));
}

__global__ void appendKernel(int T, int n, char bit, int strandCount, 
    uint8_t * length_d, int * tubeID_d, char * strands_d) {

    int strandID = blockIdx.x * blockDim.x + threadIdx.x;
    uint8_t len;

    if (strandID < strandCount) {
        if (tubeID_d[strandID] == T) {
            len = length_d[strandID];
            if (len + n > MAX_STRAND_LENGTH) {
                printf("Max strand length exceeded");
                return;
            }
            memset(&strands_d[ithBit(strandID, len)], bit, n * sizeof(char));
            length_d[strandID] = len + n;
        }
    }
}

void append(int T, int n, char bit) {
    int numBlocks = strandCount/THREADS_PER_BLOCK + 1;

    appendKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
        T,
        n, 
        bit,
        strandCount,
        length_d, 
        tubeID_d,
        strands_d);
}

__global__ void separateKernel(int T0, int T1, int T2, int i, int strandCount, 
    int * tubeID_d, char * strands_d) {

    int strandID = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (strandID < strandCount) {
        if (tubeID_d[strandID] == T0) {
            if (strands_d[ithBit(strandID, i)])
                tubeID_d[strandID] = T1;
            else
                tubeID_d[strandID] = T2;
        }
    }
}

void separate(int T0, int T1, int T2, int i) {
    int numBlocks = strandCount/THREADS_PER_BLOCK + 1;

    separateKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
        T0,
        T1,
        T2,
        i,
        strandCount,
        tubeID_d,
        strands_d);
}

__global__ void combineKernel(int T, int T1, int T2, int strandCount, int * tubeID_d) {
    int strandID = blockIdx.x * blockDim.x + threadIdx.x;

    if (strandID < strandCount) {
        if (tubeID_d[strandID] == T1 || tubeID_d[strandID] == T2) {
            tubeID_d[strandID] = T;
        }
    }
}

void combine(int T, int T1, int T2) {
    int numBlocks = strandCount/THREADS_PER_BLOCK + 1;

    combineKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
        T, 
        T1, 
        T2,
        strandCount,
        tubeID_d);

}


__global__ void setKernel(int T, int i, int strandCount,
    int * tubeID_d, char * strands_d) {

    int strandID = blockIdx.x * blockDim.x + threadIdx.x;

    if (strandID < strandCount) {
        if (tubeID_d[strandID] == T)
            strands_d[ithBit(strandID, i)] = 1;
    }
}

void set(int T, int i) {
    int numBlocks = strandCount/THREADS_PER_BLOCK + 1;

    setKernel<<<numBlocks, THREADS_PER_BLOCK>>> (
        T,
        i,
        strandCount,
        tubeID_d,
        strands_d);
}

__global__ void clearKernel(int T, int i, int strandCount,
    int * tubeID_d, char * strands_d) {
        
    int strandID = blockIdx.x * blockDim.x + threadIdx.x;

    if (strandID < strandCount) {
        if (tubeID_d[strandID] == T)
            strands_d[ithBit(strandID, i)] = 0;
    }
}

void clear(int T, int i) {
    int numBlocks = strandCount/THREADS_PER_BLOCK + 1;

    clearKernel<<<numBlocks, THREADS_PER_BLOCK>>> (
        T,
        i,
        strandCount,
        tubeID_d,
        strands_d);
}

int detect(int T) {
    int strandID;

    checkCudaErrors(hipMemcpy(tubeID_h, tubeID_d, tubeIDSize, hipMemcpyDeviceToHost));
    for (strandID = 0; strandID < strandCount; strandID++) {
        if (tubeID_h[strandID] == T)
            return strandID;
    }
    return -1;
}

__global__ void discardKernel(int T, int strandCount, int * tubeID_d) {
    int strandID = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (strandID < strandCount) {
        if (tubeID_d[strandID] == T)
            tubeID_d[strandID] = -1;
    }
}

void discard(int T) {
    int numBlocks = strandCount/THREADS_PER_BLOCK + 1;

    discardKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
        T,
        strandCount,
        tubeID_d);
}

void twoToN(int T, int N) {
    int i;

    checkCudaErrors(hipMemcpy(&tubeID_d[strandCount], &T, sizeof(int), hipMemcpyHostToDevice));
    strandCount++;
    for (i = 0; i < N; i++) {
        amplify(T, T+1);
        append(T, 1, 0);
        append(T+1, 1, 1);
        combine(T, T, T+1);
    }
}

void pascalRow(int T, int N) {
    int i, j;

    checkCudaErrors(hipMemcpy(&tubeID_d[strandCount], &T, sizeof(int), hipMemcpyHostToDevice));
    strandCount++;
    for (i = 1; i <= N; i++) {
        for (j = 0; j <= i; j++) {
            amplify(T+j, T+j+N);
            append(T+j, 1, 0);
            append(T+j+N, 1, 1);
        }
        for (j = 1; j <= i+1; j++) 
            combine(T+j, T+j, T+j-1+N);
    }   
    discard(N+1);
}
